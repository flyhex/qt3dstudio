#include "hip/hip_runtime.h"
/****************************************************************************
**
** Copyright (C) 2017 The Qt Company Ltd.
** Contact: https://www.qt.io/licensing/
**
** This file is part of Qt 3D Studio.
**
** $QT_BEGIN_LICENSE:GPL$
** Commercial License Usage
** Licensees holding valid commercial Qt licenses may use this file in
** accordance with the commercial license agreement provided with the
** Software or, alternatively, in accordance with the terms contained in
** a written agreement between you and The Qt Company. For licensing terms
** and conditions see https://www.qt.io/terms-conditions. For further
** information use the contact form at https://www.qt.io/contact-us.
**
** GNU General Public License Usage
** Alternatively, this file may be used under the terms of the GNU
** General Public License version 3 or (at your option) any later version
** approved by the KDE Free Qt Foundation. The licenses are as published by
** the Free Software Foundation and appearing in the file LICENSE.GPL3
** included in the packaging of this file. Please review the following
** information to ensure the GNU General Public License requirements will
** be met: https://www.gnu.org/licenses/gpl-3.0.html.
**
** $QT_END_LICENSE$
**
****************************************************************************/

#if defined (_PLATFORM_USE_EGL)
#include <GLES31/gl31.h>
#include <GLES31/gl2ext.h>
#endif

#include "CUDABSDFMipmap.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"
#include <iostream>

using namespace nv;
using namespace nv::render;
__host__ void jerror1(hipError_t error)
{
   static  int i = 0;
   ++i;
}
#ifdef _DEBUG
#define CHECK_AND_HANDLE_CUDA_ERROR(func)													\
		func;																			\
		{																				\
			hipError_t error = hipGetLastError();												\
			if ( error != hipSuccess )													\
			{																			\
				printf("%s\n", hipGetErrorString(error));								\
                jerror1(error);\
				NV_ASSERT( false );														\
			}																			\
		}
#else
#define CHECK_AND_HANDLE_CUDA_ERROR(func)													\
func;
#endif

__device__ inline int wrapMod( int a, int base )
{
	int ret = a % base;
	if (ret < 0 ) ret += base;
	return ret;
}

__device__ inline void getWrappedCoords( int &sX, int &sY, int width, int height )
{
	if (sY < 0) { sX -= width >> 1; sY = -sY; }
	if (sY >= height) { sX += width >> 1; sY = height - sY; }
	sX = wrapMod( sX, width );
	sY = wrapMod( sY, height );
}

__device__ void decodeToFloat( void *inPtr, NVU32 byteOfs, float *outPtr, NVRenderTextureFormats::Enum inFmt, unsigned int numberOfComponent )
{
	outPtr[0] = 0.0f;	outPtr[1] = 0.0f;	outPtr[2] = 0.0f;	outPtr[3] = 0.0f;
	NVU8 *src = reinterpret_cast<NVU8 *>(inPtr);
	//float divisor;		// If we want to support RGBD?
	switch(inFmt)
	{
	case NVRenderTextureFormats::Alpha8:
		outPtr[0] = ((float)src[byteOfs]) / 255.0f;
		break;

	case NVRenderTextureFormats::Luminance8:
	case NVRenderTextureFormats::LuminanceAlpha8:
	case NVRenderTextureFormats::R8:
	case NVRenderTextureFormats::RG8:
	case NVRenderTextureFormats::RGB8:
	case NVRenderTextureFormats::RGBA8:
	case NVRenderTextureFormats::SRGB8:
	case NVRenderTextureFormats::SRGB8A8:
		// NOTE : RGBD Hack here for reference.  Not meant for installation.
		//divisor = (NVRenderTextureFormats::getSizeofFormat(inFmt) == 4) ? ((float)src[byteOfs+3]) / 255.0f : 1.0f;
		for ( NVU32 i = 0; i < numberOfComponent; ++i )
		{
			float val = ((float)src[byteOfs + i]) / 255.0f;
			outPtr[i] = (i < 3) ? powf(val, 0.4545454545f) : val;
			// Assuming RGBA8 actually means RGBD (which is stupid, I know)
			//if ( NVRenderTextureFormats::getSizeofFormat(inFmt) == 4 ) { outPtr[i] /= divisor; }
		}
		//outPtr[3] = divisor;
		break;

	case NVRenderTextureFormats::RGBA32F:
		outPtr[0] = reinterpret_cast<float *>(src+byteOfs)[0];
		outPtr[1] = reinterpret_cast<float *>(src+byteOfs)[1];
		outPtr[2] = reinterpret_cast<float *>(src+byteOfs)[2];
		outPtr[3] = reinterpret_cast<float *>(src+byteOfs)[3];
		break;
	case NVRenderTextureFormats::RGB32F:
		outPtr[0] = reinterpret_cast<float *>(src+byteOfs)[0];
		outPtr[1] = reinterpret_cast<float *>(src+byteOfs)[1];
		outPtr[2] = reinterpret_cast<float *>(src+byteOfs)[2];
		break;

	case NVRenderTextureFormats::RGBA16F:
        /*
		for ( NVU32 i = 0; i < 4; ++i )
		{
			// NOTE : This only works on the assumption that we don't have any denormals, Infs or NaNs.
			// Every pixel in our source image should be "regular"
			NVU16 h = reinterpret_cast<NVU16 *>(src + byteOfs)[i];
			NVU32 sign = (h & 0x8000) << 16;
			NVU32 exponent = (((((h & 0x7c00) >> 10) - 15) + 127) << 23);
			NVU32 mantissa =  ((h & 0x3ff) << 13);
			NVU32 result = sign | exponent | mantissa;
					
			if (h == 0 || h == 0x8000) { result = 0; }	// Special case for zero and negative zero
			memcpy( reinterpret_cast<NVU32 *>(outPtr) + i, &result, 4 );
		}*/

		for ( NVU32 i = 0; i < 2; i++ )
		{
			// NOTE : This only works on the assumption that we don't have any denormals, Infs or NaNs.
			// Every pixel in our source image should be "regular"

			NVU32 h1 = reinterpret_cast<NVU32 *>(src + byteOfs)[i];

			for ( NVU8 j = 0; j < 2; j++ ) 
			{
				NVU16 h = (h1 & (0x0000FFFF << j*16 )) >> j*16;
				NVU32 sign = (h & 0x8000) << 16;
				NVU32 exponent = (((((h & 0x7c00) >> 10) - 15) + 127) << 23);
				NVU32 mantissa =  ((h & 0x3ff) << 13);
				NVU32 result = sign | exponent | mantissa;
					
				if (h == 0 || h == 0x8000) { result = 0; }	// Special case for zero and negative zero
				memcpy( reinterpret_cast<NVU32 *>(outPtr) + i*2 + j, &result, 4 );
			}
		}
		break;

	case NVRenderTextureFormats::R11G11B10:
		// place holder
		NV_ASSERT( false );
		break;

	default:
		outPtr[0] = 0.0f;
		outPtr[1] = 0.0f;
		outPtr[2] = 0.0f;
		outPtr[3] = 0.0f;
		break;
	}
}

void __device__ encodeToPixel( float *inPtr, void *outPtr, NVU32 byteOfs, NVRenderTextureFormats::Enum inFmt, unsigned int noOfComponent )
{
	NVU8 *dest = reinterpret_cast<NVU8 *>(outPtr);
	switch(inFmt)
	{
	case NVRenderTextureFormats::Alpha8:
		dest[byteOfs] = NVU8( inPtr[0] * 255.0f );
		break;

	case NVRenderTextureFormats::Luminance8:
	case NVRenderTextureFormats::LuminanceAlpha8:
	case NVRenderTextureFormats::R8:
	case NVRenderTextureFormats::RG8:
	case NVRenderTextureFormats::RGB8:
	case NVRenderTextureFormats::RGBA8:
	case NVRenderTextureFormats::SRGB8:
	case NVRenderTextureFormats::SRGB8A8:
		for ( NVU32 i = 0; i < noOfComponent; ++i )
		{
			inPtr[i] = (inPtr[i] > 1.0f) ? 1.0f : inPtr[i];
			if (i < 3)
				dest[byteOfs+i] = NVU8( powf( inPtr[i], 2.2f ) * 255.0f);
			else
				dest[byteOfs+i] = NVU8( inPtr[i] * 255.0f );
		}
		break;

	case NVRenderTextureFormats::RGBA32F:
		reinterpret_cast<float *>(dest+byteOfs)[0] = inPtr[0];
		reinterpret_cast<float *>(dest+byteOfs)[1] = inPtr[1];
		reinterpret_cast<float *>(dest+byteOfs)[2] = inPtr[2];
		reinterpret_cast<float *>(dest+byteOfs)[3] = inPtr[3];
		break;
	case NVRenderTextureFormats::RGB32F:
		reinterpret_cast<float *>(dest+byteOfs)[0] = inPtr[0];
		reinterpret_cast<float *>(dest+byteOfs)[1] = inPtr[1];
		reinterpret_cast<float *>(dest+byteOfs)[2] = inPtr[2];
		break;

	case NVRenderTextureFormats::RGBA16F:
		for ( NVU32 i = 0; i < 4; ++i )
		{
			// NOTE : This also has the limitation of not handling  infs, NaNs and denormals, but it should be
			// sufficient for our purposes.
			if (inPtr[i] > 65519.0f) { inPtr[i] = 65519.0f; }
			if (fabs(inPtr[i]) < 6.10352E-5f) { inPtr[i] = 0.0f; }
			NVU32 f = reinterpret_cast<NVU32 *>(inPtr)[i];
			NVU32 sign = (f & 0x80000000) >> 16;
			NVI32 exponent = (f & 0x7f800000) >> 23;
			NVU32 mantissa = (f >> 13) & 0x3ff;
			exponent = exponent - 112;
			if (exponent > 31) { exponent = 31; }
			if (exponent < 0) { exponent = 0; }
			exponent = exponent << 10;
			reinterpret_cast<NVU16 *>(dest + byteOfs)[i] = NVU16(sign | exponent | mantissa);
		}
		break;

	case NVRenderTextureFormats::R11G11B10:
		// place holder
		NV_ASSERT( false );
		break;

	default:
		dest[byteOfs] = 0;
		dest[byteOfs+1] = 0;
		dest[byteOfs+2] = 0;
		dest[byteOfs+3] = 0;
		break;
	}
}

void __global__ Convert3To4Component( hipTextureObject_t tex, float *d_outBuffer, Q3DStudio::INT32 dpitch, Q3DStudio::INT32 width, Q3DStudio::INT32 height )
{
	float *dest = d_outBuffer;

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if ( x >= width || y >= height ) 
		return;
	int inX = x * 3;
	int outX = x * 4;
	dest[outX + y * width * 4] = tex2D<float>(tex, inX, y);
	dest[outX + y * width * 4 + 1] = tex2D<float>(tex, inX + 1, y);
	dest[outX + y * width * 4 + 2] = tex2D<float>(tex, inX + 2, y);
	dest[outX + y * width * 4 + 3] = 255 * 255;
}

void __global__ ConvertData( void* d_InBuffer, NVRenderTextureFormats::Enum inFmt, int inSizeOfFormat, int inNoOfComponent, int inPitch, 
                             void* d_OutBuffer, NVRenderTextureFormats::Enum outFmt, int outSizeOfFormat, int outNoOfComponent, int outPitch, int width, int height )
{

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if ( x >= width || y >= height ) 
		return;
	float values[4];

	decodeToFloat( d_InBuffer, (inPitch * y) + (x * inSizeOfFormat), values, inFmt, inNoOfComponent );
	encodeToPixel( values, d_OutBuffer, (outPitch * y) + (x * outSizeOfFormat), outFmt, outSizeOfFormat );
}

void __global__ CreateBsdfMipLevel( hipTextureObject_t tex, void *d_curBuffer, void *d_prevBuffer, 	Q3DStudio::INT32 pitch, Q3DStudio::INT32 width, Q3DStudio::INT32 height, 
										nv::render::NVRenderTextureFormats::Enum inFormat, unsigned int sizeOfFormat )
{
	float accumVal[4];
	//unsigned int sizeofFormat = getSizeofFormat(inFormat);
	//__shared__ float dataBlock[ ]; //(32+4) * (32+4) * 12 
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ( x >= (width > 2 ? width >> 1 : 1) || y >= (height > 2 ? height >> 1 : 1)) return;

	accumVal[0] = 0;	accumVal[1] = 0;	accumVal[2] = 0;	accumVal[3] = 0;

	for ( int sy = -2; sy <= 2; ++sy )
	{
		for ( int sx = -2; sx <= 2; ++sx )
		{
			int sampleX = sx + (x << 1);
			int sampleY = sy + (y << 1);
			//getWrappedCoords(sampleX, sampleY, width, height);
			// Cauchy filter (this is simply because it's the easiest to evaluate, and requires no complex
			// functions).
			float filterPdf = 1.f / ( 1.f + float(sx*sx + sy*sy)*2.f );
			// With FP HDR formats, we're not worried about intensity loss so much as unnecessary energy gain, 
			// whereas with LDR formats, the fear with a continuous normalization factor is that we'd lose
			// intensity and saturation as well.
			filterPdf /= sizeOfFormat >= 8 ? 4.71238898f : 4.5403446f;
			//filterPdf /= 4.5403446f;		// Discrete normalization factor
			//filterPdf /= 4.71238898f;		// Continuous normalization factor
			//float curPix[4];
			sampleX = sampleX*4;
			getWrappedCoords(sampleX, sampleY, width*4, height);
			accumVal[0] += filterPdf * tex2D<float>(tex, sampleX, sampleY);
			accumVal[1] += filterPdf * tex2D<float>(tex, sampleX + 1, sampleY);
			accumVal[2] += filterPdf * tex2D<float>(tex, sampleX + 2, sampleY);
			accumVal[3] += filterPdf * tex2D<float>(tex, sampleX + 3, sampleY);
		}
		}

	encodeToPixel(accumVal, d_curBuffer, y * pitch + x * sizeOfFormat, inFormat, sizeOfFormat);
}

struct SMipTextureData
{
	void* data;
	unsigned int dataSize;
	unsigned int mipLevel;
	unsigned int width;
	unsigned int height;
	NVRenderTextureFormats::Enum format;	
};

__host__  void CUDABSDFMipMap::Build( void* inTextureData, int inTextureDataSize, NVRenderBackend::NVRenderBackendTextureObject inTextureHandle, NVRenderTextureFormats::Enum inFormat  )
{
	m_TextureHandle = inTextureHandle;
	m_InternalFormat = inFormat;
	m_SizeOfInternalFormat = NVRenderTextureFormats::getSizeofFormat( m_InternalFormat );
	m_InternalNoOfComponent = NVRenderTextureFormats::getNumberOfComponent( m_InternalFormat );

	m_Texture2D.SetTextureData(  NVDataRef<NVU8>( (NVU8*)inTextureData, inTextureDataSize )
								, 0
								, m_Width
								, m_Height
								, inFormat
								, m_DestinationFormat );

	size_t pitch;
	float* d_inTextureData;

	hipMallocPitch(&d_inTextureData, &pitch, m_Width * m_SizeOfInternalFormat, m_Height); CHECK_AND_HANDLE_CUDA_ERROR();
	CHECK_AND_HANDLE_CUDA_ERROR( hipMemcpy2D( d_inTextureData, pitch, inTextureData, m_Width * m_SizeOfInternalFormat, m_Width * m_SizeOfInternalFormat, m_Height, hipMemcpyHostToDevice ) );
	{
		dim3 blockDim(16, 16, 1);	
		dim3 gridDim(ceil(m_Width / 16.0f), ceil(m_Height / 16.0f) ,1 );

		//std::cerr << "if= " << m_InternalFormat << " sizeOut= " << m_SizeOfInternalFormat << " numOfIntComp" << m_InternalNoOfComponent << " pitch= " << pitch << " destFormat= " << m_DestinationFormat << " sizeFormat= " << m_SizeOfFormat << " numOfComp= " << m_NoOfComponent << " Pitch0=" << m_Pitches[0] << std::endl;
		//NVLogWarn("cuda", "%i %i %i %i %i %i %i %i\n",(int)m_InternalFormat ,m_SizeOfInternalFormat ,m_InternalNoOfComponent , pitch, (int)m_DestinationFormat, m_SizeOfFormat, m_NoOfComponent ,m_Pitches[0]);
        	ConvertData<<<gridDim, blockDim>>>( d_inTextureData, m_InternalFormat, m_SizeOfInternalFormat, m_InternalNoOfComponent, pitch,
                        md_MipMapsData[0], m_DestinationFormat, m_SizeOfFormat, m_NoOfComponent, m_Pitches[0], m_Width, m_Height );
	}
	hipFree(d_inTextureData);

	int curWidth  = m_Width;
	int curHeight = m_Height;

	hipTextureObject_t* tex;
	tex = new hipTextureObject_t[m_MaxMipMapLevel];
	for ( int idx = 1; idx <= m_MaxMipMapLevel; ++idx )
	{
		tex[idx-1] = -1;
		dim3 blockDim(16, 16, 1);	
		dim3 gridDim(ceil(curWidth / 32.0f), ceil(curHeight / 32.0f) ,1 );

		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.res.pitch2D.desc.f = hipChannelFormatKindFloat;
		resDesc.res.pitch2D.desc.x = m_SizeOfFormat / m_NoOfComponent * 8; // bits per channel
		resDesc.resType = hipResourceTypePitch2D;
		resDesc.res.pitch2D.devPtr = (char*)(md_MipMapsData[idx-1]);
		resDesc.res.pitch2D.height = curHeight;
		resDesc.res.pitch2D.width  = curWidth * m_NoOfComponent;
		resDesc.res.pitch2D.pitchInBytes  = m_Pitches[idx-1];// aligned to texturePitchAlignment

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.addressMode[0] = hipAddressModeWrap;
		texDesc.addressMode[1] = hipAddressModeWrap;
		texDesc.readMode = hipReadModeElementType;
		//texDesc.normalizedCoords = 1;

		
		CHECK_AND_HANDLE_CUDA_ERROR( hipCreateTextureObject( &tex[idx-1], &resDesc, &texDesc, NULL ) );
		CreateBsdfMipLevel<<<gridDim, blockDim>>>( tex[idx-1], (reinterpret_cast<NVU8 *>(md_MipMapsData[idx])), (reinterpret_cast<NVU8 *>(md_MipMapsData[idx-1])), m_Pitches[idx], curWidth, curHeight, m_DestinationFormat, m_SizeOfFormat );

		curWidth  = curWidth > 2 ? curWidth >> 1 : 1;
		curHeight = curHeight > 2 ? curHeight >> 1 : 1;
	}

	CHECK_AND_HANDLE_CUDA_ERROR( hipDeviceSynchronize(); )
	BindTexture();
	TransferTexture();
	for (int idx = 0; idx < m_MaxMipMapLevel;++idx )
		hipDestroyTextureObject(tex[idx]);
//	CHECK_AND_HANDLE_CUDA_ERROR( hipDeviceReset(); )
	CHECK_AND_HANDLE_CUDA_ERROR( hipDeviceSynchronize(); )

    //NV_FREE( m_Foundation.getAllocator(), inTextureData );

}
